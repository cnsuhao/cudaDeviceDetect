#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

int main (int argc,char** argv)
{
	int deviceCount;   //the numbers of cudaDevive
	hipError_t cudaStatus;  //catch the error
	
	cudaStatus = hipGetDeviceCount(&deviceCount);      //get the numbers of the cudaDevice

	if(cudaStatus != hipSuccess){
		printf("get cudaDevice numbers Error %d\n",cudaStatus);
		return false;
	}

	if(0 == deviceCount){
		printf("There is no device supporting CUDA\n");
	}

	int dev;

	for(dev = 0; dev < deviceCount; ++dev ){
		hipDeviceProp_t deviceProp;
		cudaStatus = hipGetDeviceProperties(&deviceProp,dev);

		if(cudaStatus != hipSuccess){
			printf("Get cudaDeviceProperties error: %d",cudaStatus);
			return false;
		}

		if( 0 == dev ){
			if(deviceProp.major == 9999 && deviceProp.minor == 9999)
				printf("This is no device supporting CUDA.\n");
			else if(1 == deviceCount)
				printf("There is 1 device supporting CUDA.\n");
			else
				printf("There are %d devices supporting CUDA\n",deviceCount);
		}
		printf("\nDevice %d:\"%s\"\n", dev,deviceProp.name);
		printf("	Major revision number:				%d\n",
					deviceProp.major);
		printf("	Minor revision number:				%d\n",
			        deviceProp.minor);

		printf("	Total amount of global memory:			%u bytes\n",
					deviceProp.totalGlobalMem);


#if CUDART_VERSION >= 2000
	printf("	Number of multiprocessors			%d\n",
				deviceProp.multiProcessorCount);						//numbers of SM
	printf("	Number of cores:				%d\n",
				8*deviceProp.multiProcessorCount);                      //numbers of SP
#endif


		printf("	Total amount of constant memory:		%u bytes\n",
					deviceProp.totalConstMem);

		printf("	Total amount of shared memory per block:	%u bytes\n",
					deviceProp.sharedMemPerBlock);

		printf("	Total number of registers available per block:	%d\n",
					deviceProp.regsPerBlock);

		printf("	Maximum sizes of each dimemsion of a block:	%d x %d x %d\n",
			deviceProp.maxThreadsDim[0],
			deviceProp.maxThreadsDim[1],
			deviceProp.maxThreadsDim[2]);

		printf("	Maximum sizes of each dimemsion of a grib:	%d x %d x %d\n",
			deviceProp.maxGridSize[0],
			deviceProp.maxGridSize[1],
			deviceProp.maxGridSize[2]);


		printf("	Maximum memproy pitch:				%u bytes\n",
					deviceProp.memPitch);

		printf("	Texture alignment:				%u bytes\n",
					deviceProp.textureAlignment);

		printf("	Clock rate:					%.2f GHz\n",
					deviceProp.clockRate * 1e-6f);
#if CUDART_VERSION >= 2000
	printf("	Concurrent copy and execution:			%s\n",
		deviceProp.deviceOverlap ? "Yes" : "No");
#endif
	}
	printf("\nTest PASSED\n");
	
	return true;
}
